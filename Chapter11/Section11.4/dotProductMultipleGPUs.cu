#include "hip/hip_runtime.h"
#include<iostream>
#include "../../common/book.h"

#define imin(a,b) (a < b ? a : b)

const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

struct DataStruct {
    int deviceID;
    int size;
    float *a;
    float *b;
    float returnValue;
};

__global__ void dot(int size, float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;

   float temp = 0;
   while (tid < size){
       temp += a[tid] * b[tid];
       tid += blockDim.x * gridDim.x;
   }

   // set the cache values
   cache[cacheIndex] = temp;

   // synchronize threads in this block
   __syncthreads();

   // for reductions, threadsPerBlock must be a power of 2
   // because of the following code
   int i = blockDim.x/2;
   while (i != 0){
       if (cacheIndex < i)
           cache[cacheIndex] += cache[cacheIndex + i];
       __syncthreads();
       i /= 2;
   }

   if (cacheIndex == 0)
       c[blockIdx.x] = cache[0];
}

void* routine(void *pvoidData) {
    DataStruct *data = (DataStruct*)pvoidData;
    
    if (data->deviceID != 0) {
        hipSetDevice(data->deviceID);
        hipSetDeviceFlags(hipDeviceMapHost);
    }

    int size = data->size;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    // allocate memory on the CPU side
    a = data->a;
    b = data->b;
    partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

    // allocate the memory on the GPU
    hipHostGetDevicePointer(&dev_a, a, 0);
    hipHostGetDevicePointer(&dev_b, b, 0);
    hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float));

    // launch kernel
    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

   // copy the array 'c' back from the GPU to the CPU
   hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost);
 
   //finish up on the CPU side
   c = 0;
   for (int i = 0; i < blocksPerGrid;i++){
       c += partial_c[i];
   }

   // free memory on the GPU side
   hipFree(dev_partial_c);

   // free memory on the CPU side
   free(partial_c);
   
   data->returnValue = c;
   return 0;
}

int main(void) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < 2) {
        std::cout << "We need at least two compute 1.0 or greater devices, but only found " << deviceCount << std::endl;
        return 0;
    }

    hipDeviceProp_t prop;
    for (int i = 0; i < 2; i++){
        hipGetDeviceProperties(&prop, i);
        if (prop.canMapHostMemory != 1) {
            std::cout << "Device " << i << " cannot map memory." << std::endl;
            return 0;
        }
    }    

    float *a, *b;
    hipSetDevice(0);
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc((void**)&a, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped);
    hipHostAlloc((void**)&b, N * sizeof(float), hipHostMallocWriteCombined | hipHostMallocPortable | hipHostMallocMapped);    

    // fill in the host memory with data
    for (int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i * 2;
    }

    DataStruct data[2];
    data[0].deviceID = 0;
    data[0].size = N / 2;
    data[0].a = a;
    data[0].b = b;

    data[1].deviceID = 1;
    data[1].size = N / 2;
    data[1].a = a + N / 2;
    data[1].b = b + N / 2;

    CUTThread thread = start_thread(routine, &(data[1]));
    routine(&(data[0]));

    end_thread(thread);    

    // free memory on the CPU side
    hipHostFree(a);
    hipHostFree(b);

    std::cout << "Value calculated: " << data[0].returnValue + data[1].returnValue << std::endl;

    return 0;
}
