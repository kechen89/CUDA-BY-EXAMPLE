#include "hip/hip_runtime.h"
#include<iostream>

#define imin(a,b) (a < b ? a : b)

const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int size, float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;

   float temp = 0;
   while (tid < size){
       temp += a[tid] * b[tid];
       tid += blockDim.x * gridDim.x;
   }

   // set the cache values
   cache[cacheIndex] = temp;

   // synchronize threads in this block
   __syncthreads();

   // for reductions, threadsPerBlock must be a power of 2
   // because of the following code
   int i = blockDim.x/2;
   while (i != 0){
       if (cacheIndex < i)
           cache[cacheIndex] += cache[cacheIndex + i];
       __syncthreads();
       i /= 2;
   }

   if (cacheIndex == 0)
       c[blockIdx.x] = cache[0];
}

float cuda_host_alloc_test(int size) {
    hipEvent_t start, stop;
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate memory on the CPU side
    hipHostAlloc((void**)&a, size*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);    
    hipHostAlloc((void**)&b, size*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);
    hipHostAlloc((void**)&partial_c, size*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped);

    // fill in the host memory with data
    for (int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i * 2;
    }

    hipHostGetDevicePointer(&dev_a, a, 0);
    hipHostGetDevicePointer(&dev_b, b, 0);
    hipHostGetDevicePointer(&dev_partial_c, partial_c, 0);
    
    hipEventRecord(start, 0);

    // launch kernel
    dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

    hipDeviceSynchronize(); 

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    //finish up on the CPU side
    c = 0;
    for (int i = 0; i < blocksPerGrid;i++){
        c += partial_c[i];
    }

    #define sum_squares(x) (x*(x+1)*(2*x+1)/6)
    std::cout << "Does GPU value " << c << " = " << 2 * sum_squares((float)(N - 1)) << " ? " << std::endl;

    // free memory on the CPU side
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(partial_c);

    // free events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

int main(void) {
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (prop.canMapHostMemory != 1) {
        std::cout << "Device cannot map memory." << std::endl;
        return 0;
    }

    hipSetDeviceFlags(hipDeviceMapHost);

    float elapsedTime = cuda_host_alloc_test(N);
    std::cout << "Time using hipHostAlloc: " << elapsedTime << std::endl;
}
