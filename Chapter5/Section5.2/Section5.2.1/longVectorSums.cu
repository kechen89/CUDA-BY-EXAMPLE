
#include <hip/hip_runtime.h>
#include <iostream>

#define N (33 * 1024)

__global__ void add(int *a, int *b, int *c){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N){
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // launch kernel
    add<<<128,128>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // verify that the GPU did the work we requested
    bool success = true;
    for (int i = 0; i < N; i++){
        if ((a[i] + b[i]) != c[i]){
            std::cout << "Error: " << a[i] << " + " << b[i] << " != " << c[i] << std::endl;
            success = false;
        }
    }
    if (success) std::cout << "We did it!" << std::endl;

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
