#include<iostream>

int main(void) {
    hipDeviceProp_t prop;   
    int count;

    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        std::cout << "--- General Information for device" << i << "---" << std::endl;
        std::cout << "Name:" << prop.name << std::endl;
        std::cout << "Compute capability:" << prop.major << prop.minor << std::endl;
        std::cout << "Clock rate:" << prop.clockRate << std::endl;
        std::cout << "Device copy overlap:" << std::endl;
        if (prop.deviceOverlap)
            std::cout << "Enabled" << std::endl;
        else
            std::cout << "Disabled" << std::endl;
        std::cout << "Kernel execition timeout:" << std::endl;
        if (prop.kernelExecTimeoutEnabled)
            std::cout << "Enabled" << std::endl;
        else
            std::cout << "Disabled" << std::endl;
        std::cout << "---Memory Information for device" << i << "---" << std::endl;
        std::cout << "Total global mem:" << prop.totalGlobalMem << std::endl;
        std::cout << "Total constant Mem:" << prop.totalConstMem << std::endl;
        std::cout << "Max mem pitch:" << prop.memPitch << std::endl;
        std::cout << "Texture Aligment:" << prop.textureAlignment << std::endl;
        std::cout << "--- MP Information for device" << i << "---" << std::endl;
        std::cout << "Multiprocessor count:" << prop.multiProcessorCount << std::endl;
        std::cout << "Shared mem per mp:" << prop.sharedMemPerBlock << std::endl;
        std::cout << "Registers per mp:" << prop.regsPerBlock << std::endl;
        std::cout << "Threads in warp:" << prop.warpSize << std::endl;
        std::cout << "Max threads per block:" << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Max threads dimensions:" << prop.maxThreadsDim[0] << prop.maxThreadsDim[1] << prop.maxThreadsDim[2] << std::endl;
        std::cout << "Max grid dimensions:" << prop.maxGridSize[0] << prop.maxGridSize[1] << prop.maxGridSize[2] << std::endl;      
    } 

}
