
#include <hip/hip_runtime.h>
#include<iostream>

int main(void) {
    hipDeviceProp_t prop;   
    int count;

    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        std::cout << "General Information for device" << i << std::endl;
        std::cout << "Name:" << prop.name << std::endl;
        std::cout << "Compute capability:" << prop.major << prop.minor << std::endl;
    } 

}
