#include "hip/hip_runtime.h"
#include <iostream>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int *a, int *b, int *c){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs)/2;
    }
}

int main(){
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        std::cout << "Device will not handle overlaps, so no speed up from streams" << std::endl;

        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;

    // start the timers
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // initialize the stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // allocate page-locked memory, used to stream
    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // now loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N) {
        // copy the locked memory to the device, async
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        
        kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        // copy the data from device to locked memory
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }

    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time taken: " << elapsedTime << std::endl;

    // cleanup the streams and memory
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipStreamDestroy(stream);
    return 0;
}
