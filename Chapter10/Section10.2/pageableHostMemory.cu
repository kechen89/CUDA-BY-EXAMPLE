
#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE (10*1024*1024)

float cuda_malloc_test(int size, bool up);

int main(void) {
    float elapsedTime;
    float MB = (float)100 * SIZE * sizeof(int)/1024/1024;

    elapsedTime = cuda_malloc_test(SIZE, true);

    std::cout << "Time using cudaMalloc: " << elapsedTime << std::endl;

    std::cout << "MB/s during copy up: " << MB/(elapsedTime/1000) << std::endl;

    elapsedTime = cuda_malloc_test(SIZE, false);

    std::cout << "Time using cudaMalloc: " << elapsedTime << std::endl;

    std::cout << "MB/s during copy down: " << MB/(elapsedTime/1000) << std::endl;
}

float cuda_malloc_test(int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    a = (int*)malloc(size * sizeof(*a));
    hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

    hipEventRecord(start, 0);
    for (int i = 0; i < 100; i++){
        if (up)
            hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
        else
            hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime,start, stop);
    
    free(a);
    hipFree(dev_a);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

